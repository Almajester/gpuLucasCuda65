#include "hip/hip_runtime.h"
/**
* cufftCallbacks.cu
*
* A. Thall
* Alma College
* 5/28/2014
*
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hipfft/hipfftXt.h> 
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

// Create ThreadsPerBlock constant
const int T_PER_B = 1024;
const int SIGNAL_SIZE = 65536;

// Complex data type
typedef hipfftDoubleComplex dbComplex;
typedef hipfftDoubleReal dbReal;
#define CUFFT_TYPEFORWARD HIPFFT_D2Z
#define CUFFT_TYPEINVERSE HIPFFT_Z2D
#define CUFFT_EXECFORWARD hipfftExecD2Z
#define CUFFT_EXECINVERSE hipfftExecZ2D

void callbackTest(int signalSize);
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess) {
		fprintf(stderr, "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
	}
	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
		fprintf(stderr, "There is no device supporting CUDA\n");
	else
		fprintf(stderr, "Found %d CUDA Capable device(s)\n", deviceCount);

	int dev;
	for (dev = 0; dev < deviceCount; ++dev) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);

		fprintf(stderr, "\nDevice %d: \"%s\"\n", dev, deviceProp.name);
	}
	fprintf(stderr, "\n and deviceID of max GFLOPS device is %d\n", gpuGetMaxGflopsDeviceId());
	fprintf(stderr, "but we're going to use device 0 by default.\n");
	hipSetDevice(0);//gpuGetMaxGflopsDeviceId());

	printf("  NUM_BLOCKS = %d, T_PER_B = %d\n", SIGNAL_SIZE/T_PER_B, T_PER_B);

	callbackTest(SIGNAL_SIZE);

	hipDeviceReset();	
	exit(0);
}

/**
 * HERE BEGINS THE HOST AND KERNEL CODE TO SUPPORT THE APPLICATION
 *   NOTE:  some changed, moved to IrrBaseBalanced11.cu
 */

// Complex pointwise multiplication...divide by signal size to get normalized
static __global__ void dbcPointwiseSqr(dbComplex* cval, int size)
{
	dbComplex c, temp;
	const int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid < size) {
		temp = cval[tid];
		c.y = 2.0*temp.x*temp.y/SIGNAL_SIZE;
		//c.x = (temp.x + temp.y)*(temp.x - temp.y);  xxAT ??
		c.x = (temp.x*temp.x - temp.y*temp.y)/SIGNAL_SIZE;
		cval[tid] = c;
	}
} 


/** 
 * LET'S do the above with a cuFFT callback using the new CUDA6.5
 * callback protocols
 */

__device__ void dbcPointwiseSqrCB(void *dataOut, size_t offset,
									  dbComplex element, void *callerInfo,
									  void *sharedPointer) {
   dbComplex temp = element;
   element.y = 2.0*temp.x*temp.y/SIGNAL_SIZE;
   element.x = (temp.x*temp.x - temp.y*temp.y)/SIGNAL_SIZE;
   ((dbComplex *)dataOut)[offset] = element;
}

__device__ hipfftCallbackStoreZ csquareCBptr = dbcPointwiseSqrCB;

__device__ dbComplex dbcPointwiseSqrLoadCB(void *dataIn, size_t offset,
										  void *callerInfo, void *sharedPointer) {
   dbComplex ret, element = ((dbComplex *) dataIn)[offset];
   ret.x = element.x*element.x - element.y*element.y;
   ret.y = 2.0*element.x*element.y;
   return ret;
}

__device__ hipfftCallbackLoadZ csquareLOADCBptr = (hipfftCallbackLoadZ) dbcPointwiseSqrLoadCB;

// load values of int array into double array for FFT.  Low-order 2 bytes go in lowest numbered
//     position in dArr
static __global__ void loadValue4ToFFTarray(double *dArr, int size) {

	const int tid = blockIdx.x*blockDim.x + threadIdx.x;

	if (tid < 2)
		dArr[tid] = 1.0;
	else
		dArr[tid] = 0.0;
}

/**
* callbackTest() -- test the cufftCallback functionality in CUDA 6.5 pre-release
*/
void callbackTest(int signalSize) {

	// We assume throughout that signalSize is divisible by T_PER_B
	const int numBlocks = signalSize/T_PER_B;
	const int numFFTblocks = (signalSize/2 + 1)/T_PER_B + 1;

	// Allocate device memory for signal
	dbReal *d_signal;
	dbComplex *z_signal;

	int d_size = sizeof(dbReal)*signalSize;
	int z_size = sizeof(dbComplex)*(signalSize/2 + 1);

	checkCudaErrors(hipMallocManaged(&d_signal, d_size));
	checkCudaErrors(hipMallocManaged(&z_signal, z_size));

	// allocate device memory for DWT weights and base values
	// CUFFT plan
	hipfftHandle plan1, plan2;
	checkCudaErrors(hipfftPlan1d(&plan1, signalSize, CUFFT_TYPEFORWARD, 1));
	checkCudaErrors(hipfftPlan1d(&plan2, signalSize, CUFFT_TYPEINVERSE, 1));

	/** xxAT ** get callbackPtr for fftCallback squaring */

	hipfftCallbackStoreZ hostCopyPtr;
   	checkCudaErrors(hipMemcpyFromSymbol(&hostCopyPtr, HIP_SYMBOL(csquareCBptr),
										 sizeof(hostCopyPtr)));
	hipfftCallbackStoreZ pters[1];
	pters[0] = hostCopyPtr;
	fprintf(stderr, "The host pointer to the device function is %d\n", hostCopyPtr);
	fflush(stderr);
   	checkCudaErrors(hipfftXtSetCallback(plan1, (void **) pters,
	   								   HIPFFT_CB_ST_COMPLEX_DOUBLE, NULL));
   	/*
	hipfftCallbackLoadZ hostCopyPtr;
   	checkCudaErrors(hipMemcpyFromSymbol(&hostCopyPtr, HIP_SYMBOL(csquareLOADCBptr),
										 sizeof(hostCopyPtr)));
	hipfftCallbackLoadZ pters[1];
	pters[0] = hostCopyPtr;
	fprintf(stderr, "The host pointer to the device function is %d\n", hostCopyPtr);
	fflush(stderr);
	//   	checkCudaErrors(hipfftXtSetCallback(plan2, (void **) pters,
	//   								   HIPFFT_CB_LD_COMPLEX_DOUBLE, NULL));
	*/
	// load the int array to the doubles for FFT
	// This is already balanced, and already multiplied by a_0 = 1 for DWT
	loadValue4ToFFTarray<<<numBlocks, T_PER_B>>>(d_signal, signalSize);
	getLastCudaError("Kernel execution failed [ loadValue4ToFFTarray ]");

	hipDeviceSynchronize();

	for (int i = 0; i < 20; i++)
		fprintf(stderr, "%f ", d_signal[i]);
	fprintf(stderr, "\n\n");
	fflush(stderr);

	fprintf(stderr, "starting loop\n");
	fflush(stderr);
	// Loop M-2 times

	for (int iter = 0; iter < 4; iter++) {

		// Transform signal
		checkCudaErrors(CUFFT_EXECFORWARD(plan1, (dbReal *)d_signal, (dbComplex *)z_signal));
		getLastCudaError("Kernel execution failed [ CUFFT_EXECFORWARD ]");
		hipDeviceSynchronize();
		for (int z = 0; z < 20; z++)
			fprintf(stderr, "(%f,%f) ", z_signal[z].x, z_signal[z].y);
		fprintf(stderr, "\n\n");
		fflush(stderr);


		//fprintf(stderr, "Completed one forward fft at iteration %d\n", iter);
		// fflush(stderr);
		// Multiply the coefficients componentwise
   		//dbcPointwiseSqr<<<numFFTblocks, T_PER_B>>>(z_signal, signalSize/2 + 1);
   		//getLastCudaError("Kernel execution failed [ ComplexPointwiseSqr ]");

   		hipDeviceSynchronize();
		// Transform signal back
		checkCudaErrors(CUFFT_EXECINVERSE(plan2, (dbComplex *)z_signal, (dbReal *)d_signal));
		getLastCudaError("Kernel execution failed [ CUFFT_EXECINVERSE ]");

		hipDeviceSynchronize();
		for (int i = 0; i < 20; i++)
			fprintf(stderr, "%f ", d_signal[i]);
		fprintf(stderr, "\n\n");
		fflush(stderr);
	}

	fprintf(stderr, "\nTests completed.\n");

	//Destroy CUFFT context
	checkCudaErrors(hipfftDestroy(plan1));
	checkCudaErrors(hipfftDestroy(plan2));

	// cleanup memory
	checkCudaErrors(hipFree(d_signal));
	checkCudaErrors(hipFree(z_signal));
}
